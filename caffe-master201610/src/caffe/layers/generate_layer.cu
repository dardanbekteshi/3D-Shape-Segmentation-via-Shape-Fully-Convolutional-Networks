#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/generate_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {
template <typename Dtype>
__global__ void GenerateForward(const int n, const Dtype* in1,const Dtype* in2, Dtype* out,
const int num,const int channels,const int height,const int next_kernel_w){
	CUDA_KERNEL_LOOP(index,n){
		const int offset_width = index % next_kernel_w;
		const int offset_height = (index / next_kernel_w) % height;
		const int offset_channel = (index / next_kernel_w / height) % channels;
		const int offset_n = index / next_kernel_w / height /channels;
		int offset_b1 = (offset_n*channels + offset_channel) * height;
		if(offset_width > 0){
			//
			int offset_b2 = offset_n*height*(next_kernel_w - 1) + offset_height * (next_kernel_w - 1) + (offset_width - 1);
			int tmp_offset = in2[offset_b2];
			out[index] = in1[offset_b1 + tmp_offset];
		}else{
			out[index] = in1[offset_b1 + offset_height];
		}
	}
}
template <typename Dtype>
void GenerateLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
	const vector<Blob<Dtype>*>& top) {
	const Dtype* bottom_data = bottom[0]->gpu_data();
	const Dtype* index_data = bottom[1]->gpu_data();
	Dtype* top_data = top[0]->mutable_gpu_data();
	const int count = top[0]->count();
	//c*h*w sequence
	GenerateForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
	count, bottom_data, index_data, top_data, bottom[0]->num(), channel_, height_, next_kernel_w_);
	CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void GenerateBackward(const int n, const Dtype* in, Dtype* out, const int next_kernel_w){
	CUDA_KERNEL_LOOP(index,n){
        out[index] = in[index * next_kernel_w];
}
}

template <typename Dtype>
void GenerateLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	if (propagate_down[0]) {
		const Dtype* top_diff = top[0]->gpu_diff();
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
		const int count = bottom[0]->count();
		caffe_gpu_set(bottom[1]->count(), Dtype(0), bottom[1]->mutable_gpu_diff());
		GenerateBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
     	count, top_diff, bottom_diff, next_kernel_w_);
	    CUDA_POST_KERNEL_CHECK;
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(GenerateLayer);

}  // namespace caffe
